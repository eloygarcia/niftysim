
#include <hip/hip_runtime.h>
// =========================================================================
// File:       tledContactUSProbe.cu
// Purpose:    Create instance of a rigid ultrasound probe for contact
// Package:    NiftySim: Nonlinear finite element program
// Author:     Zeike Taylor
// Language:   C++
// Created:    August 2010
// 
// Copyright (c) 2010, University of Queensland. All rights reserved.
// MedTeQ Centre
// See the LICENSE.txt file in the root folder
// 
// ztaylor@itee.uq.edu.au
// =========================================================================

#ifdef _GPU_

#include "tledContactUSProbe.h"

#include <iostream>

using namespace std;

tledContactUSProbe::tledContactUSProbe(vector<float> orig, vector<float> axis, float R, float L, vector<int> slvs, vector<float> origdisp, float radchng, int NumNodes)
{
   for (int i = 0; i < 3; i++)
   {
      Origin0[i] = orig[i];
      Axis0[i] = axis[i];
      OriginDisp[i] = origdisp[i];
   }
   memcpy(Origin,Origin0,sizeof(float)*3);
   memcpy(Axis,Axis0,sizeof(float)*3);
   Radius0 = Radius = R;
   Length0 = Length = L;
   slvNodes = slvs;
   RadiusChng = radchng;
   
   // Allocate device variable
   
   tledGPUContactUSProbe l_Prb;
   l_Prb.Origin.x = Origin[0];
   l_Prb.Origin.y = Origin[1];
   l_Prb.Origin.z = Origin[2];
   l_Prb.Origin.w = Radius;
   l_Prb.Axis.x = Axis[0];
   l_Prb.Axis.y = Origin[1];
   l_Prb.Axis.z = Origin[2];
   l_Prb.Axis.w = Radius;
   cudaMalloc((void**)&(l_Prb.SlaveMask),sizeof(int)*NumNodes);
   int* l_SlaveMask = new int[NumNodes];
   memset(l_SlaveMask,0,sizeof(int)*NumNodes);
   for (int i = 0; i < (int)slvNodes.size(); i++)
      l_SlaveMask[slvNodes[i]] = 1;
   cudaMemcpy(l_Prb.SlaveMask,l_SlaveMask,sizeof(int)*NumNodes,cudaMemcpyHostToDevice);
   
   cudaMalloc((void**)&d_Prb,sizeof(tledGPUContactUSProbe));
   
   cudaMemcpy(d_Prb,&l_Prb,sizeof(tledGPUContactUSProbe),cudaMemcpyHostToDevice);
   delete l_SlaveMask;
}

void tledContactUSProbe::ReleaseGPUMemory(tledGPUContactUSProbe *dp_prb) {
  int *dp_slaveMask = NULL;

  tledCUDAHelpers::CopyFromDevice<int*>(&dp_slaveMask, &dp_prb->SlaveMask);
  tledCheckCUDAErrors(cudaFree(dp_slaveMask));
  tledCheckCUDAErrors(cudaFree(dp_prb));
}

void tledContactUSProbe::Update(double TR)
{
   double TR2 = TR*TR;
   float Amp = (float)( TR2*(10*TR - TR2*(15 - 6*TR)) );
   // Update origin
   for (int j = 0; j < 3; j++)
      Origin[j] = Origin0[j] + OriginDisp[j]*Amp;
   // Update radius
   Radius = Radius0 + RadiusChng*Amp;
   // Update device variables
   float4 l_Origin;
   l_Origin.x = Origin[0];
   l_Origin.y = Origin[1];
   l_Origin.z = Origin[2];
   l_Origin.w = Radius;
   cudaMemcpy(&(d_Prb->Origin),&l_Origin,sizeof(float4),cudaMemcpyHostToDevice);
   float4 l_Axis;
   l_Axis.x = Axis[0];
   l_Axis.y = Axis[1];
   l_Axis.z = Axis[2];
   l_Axis.w = Length;
   cudaMemcpy(&(d_Prb->Axis),&l_Axis,sizeof(float4),cudaMemcpyHostToDevice);
}

vector<float> tledContactUSProbe::GetStartOriginV()
{
   vector<float> orig(3);
   orig[0] = Origin0[0]; orig[1] = Origin0[1]; orig[2] = Origin0[2];
   return orig;
}

vector<float> tledContactUSProbe::GetStartAxisV()
{
   vector<float> axis(3);
   axis[0] = Axis0[0]; axis[1] = Axis0[1]; axis[2] = Axis0[2];
   return axis;
}

vector<float> tledContactUSProbe::GetOriginDispV()
{
   vector<float> disp(3);
   disp[0] = OriginDisp[0]; disp[1] = OriginDisp[1]; disp[2] = OriginDisp[2];
   return disp;
}

void tledContactUSProbe::SetOriginDisp(vector<float> disp)
{
   if (disp.size() != 3)
   {
      cerr << "!!! Warning: invalid vector passed" << endl;
      return;
   }
   for (int i = 0; i < 3; i++)
      OriginDisp[i] = disp[i];
}

void tledContactUSProbe::SetStartOrigin(vector<float> orig)
{
   if (orig.size() != 3)
   {
      cerr << "!!! Warning: invalid vector passed" << endl;
      return;
   }
   for (int i = 0; i < 3; i++)
      Origin0[i] = orig[i];
}

void tledContactUSProbe::SetStartAxis(vector<float> axis)
{
   if (axis.size() != 3)
   {
      cerr << "!!! Warning: invalid vector passed" << endl;
      return;
   }
   for (int i = 0; i < 3; i++)
      Axis0[i] = axis[i];
}

#endif // _GPU_

